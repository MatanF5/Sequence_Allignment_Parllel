#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <string.h>
#include "structs.h"


__device__ const char* firstConservative[]={"NDEQ","NEQK","STA","MILV","QHRK","NHQK","FYW","HY","MILF"};

__device__ const char* secondConservative[]={"SAG","ATV","CSA","SGND","STPA","STNK","NEQHRK","NDEQHK","SNDEQK","HFY","FVLIM"};


// Comparing letters 
__device__ int compareChar(const char* s, char c)
{
	do{
		if(*s == c) return 1;
	}while (*s++);
	return 0;
}

// Checking for conservation 
__device__ int checkConservative(const char* conservative[],const int size,char ch1, char ch2)
{
	for (int i =0; i< size;i++)
    {
		if(compareChar(conservative[i],ch1) && compareChar(conservative[i],ch2))
			return 1;
	}
	return 0;
}

// Same function as strcpy in C to help with changing the mutation
__device__ char * my_strcpy(char *dest, const char *src){

	int i = 0;
	do {
		dest[i] = src[i];}
	while (src[i++] != 0);
	return dest;

}


//creating the mutation itself
__global__  void createMutant(char *arr,int SeqLength,int firstIndex,int lastIndex,int offset, char *Seq1, char *currSeq2){

	char* temp = currSeq2;

	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= SeqLength-1)
		return;
	//changing to mutant (dleteing the letters in the indexes given)
	my_strcpy(&temp[firstIndex],&temp[firstIndex+1]);
	my_strcpy(&temp[lastIndex-1],&temp[lastIndex]);

    //Formating the mutation as the given symbols
	if(Seq1[i + offset] == temp[i])
		arr[i] = '*';
	else if (checkConservative(firstConservative,9,Seq1[offset+i], temp[i]))
		arr[i] = ':';
	else if (checkConservative(secondConservative,11,Seq1[offset+i], temp[i]))
		arr[i] = '.';
	else
		arr[i] = ' ';

}

// Using gPU to implement a quicker and more efficent search for mutation and calculations
char* useGPU(int seqLength, int firstIndex, int lastIndex, int offset, char *Seq1, char* currSeq2, int tid){


	// Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

	//Using cuda stream a sequence of operations that execute on the device in the order in which they are issued by the host code
	const int num_streams = 8;
	hipStream_t stream[num_streams];
	hipStreamCreate(&stream[tid]);
	// Allocate memory on GPU to copy the data from the host
    char *d_Mutant;
	size_t size = seqLength * sizeof(char);
    err = hipMalloc((void **)&d_Mutant, size);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	hipMemset(d_Mutant,0,size);
	 // Allocate memory on GPU to copy the data from the host
    char *d_Seq1;
	size_t size_Seq1= (strlen(Seq1) + 1) * sizeof(char);
    err = hipMalloc((void **)&d_Seq1, size_Seq1);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Copy data from host to the GPU memory
    err = hipMemcpy(d_Seq1, Seq1, size_Seq1, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	// Allocate memory on GPU to copy the data from the host
    char *d_currSeq2;
	size_t size_currSeq2 = (strlen(currSeq2) + 1) * sizeof(char);
    err = hipMalloc((void **)&d_currSeq2, size_currSeq2);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy data from host to the GPU memory
    err = hipMemcpy(d_currSeq2, currSeq2, size_currSeq2, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	int threadsPerBlock = 256;
    int blocksPerGrid =(seqLength + threadsPerBlock - 1) / threadsPerBlock;
    // Using 3D array for a faster solution
	createMutant<<<blocksPerGrid, threadsPerBlock, 0 , stream[tid]>>>(d_Mutant,seqLength,firstIndex,lastIndex,offset,d_Seq1,d_currSeq2);
    // check if the mutation failed.
	err = hipGetLastError();
    if (err != hipSuccess){
        fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	char* result = (char*)malloc(seqLength*sizeof(char));
    // Copy the  result from GPU to the host memory.
    err = hipMemcpyAsync(result, d_Mutant, seqLength, hipMemcpyDeviceToHost, stream[tid]);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Free allocated memory on GPU:
    if (hipFree(d_Mutant) != hipSuccess){
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    if (hipFree(d_Seq1) != hipSuccess){
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    if (hipFree(d_currSeq2) != hipSuccess){
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	hipStreamDestroy(stream[tid]);
    return result;
}